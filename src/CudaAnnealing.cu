#include "hip/hip_runtime.h"
/*
 * CudaAnnealing.cu
 *
 *  Created on: Feb 6, 2019
 *      Author: alexander
 */

#include "Matrix.h"
#include "Spinset.h"
#include "CudaAnnealing.h"
#include <hip/hip_runtime.h>
#include <sstream>
#include <math.h>

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

CudaAnnealing::CudaAnnealing(Matrix _matrix, int _blockCount, float _minDiff) {
	minDiff = _minDiff;
	// Set pointers to null
	devSpins = NULL;
	devMat = NULL;
	devUnemptyMat = NULL;
	meanFieldMembers = NULL;
	hamiltonianMembers = NULL;
	continueIteration = NULL;
	devTemp = NULL;

	size = _matrix.getSize();
	blockSize = 512;
	blockCount = _blockCount;

	hipDeviceProp_t deviceProp;
	checkError(hipGetDeviceProperties(&deviceProp, 0), "getProp");
	blockSize = deviceProp.maxThreadsPerBlock;

	// Allocate memory for pointers at GPU
	checkError(
			hipMalloc((void**) &meanFieldMembers,
					sizeof(float) * size * blockCount), "malloc");
	hipMalloc((void**) &devMat, sizeof(float) * size * size);
	hipMalloc((void**) &devSpins, sizeof(float) * size * blockCount);
	hipMalloc((void**) &devUnemptyMat, sizeof(int) * size * (size + 1));
	hipMalloc((void**) &hamiltonianMembers, sizeof(double) * size * size);
	hipMalloc((void**) &devTemp, sizeof(float) * blockCount);
	hipMalloc((void**) &continueIteration, sizeof(bool) * _blockCount);

	// Copy model data to GPU memory
	checkError(
			hipMemcpy(devMat, _matrix.getArray(), sizeof(float) * size * size,
					hipMemcpyHostToDevice), "memcpy mat to host");
	hipMemcpy(devUnemptyMat, _matrix.getUnemptyMat(),
			sizeof(int) * size * (size + 1), hipMemcpyHostToDevice);
}

void CudaAnnealing::loadSet(Spinset set, int setIndex) {
	checkError(
			hipMemcpy(&devSpins[setIndex * size], set.getArray(),
					sizeof(float) * size, hipMemcpyHostToDevice),
			"memcpy spinset to device");
	hipMemcpy(&devTemp[setIndex], &(set.temp), sizeof(float),
			hipMemcpyHostToDevice);
}

void CudaAnnealing::freeAllocatedMemory() {
	// Free GPU memory
	hipFree(devSpins);
	hipFree(devMat);
	hipFree(meanFieldMembers);
	hipFree(devTemp);
	hipFree(devUnemptyMat);
	hipFree(hamiltonianMembers);
	hipFree(continueIteration);
}

__global__ void allocateHamiltonianMembers(float* devMat, float* devSpins,
		int setIndex, int size, double* hamiltonianMembers) {
	// Hamiltonian member assignment
	int i;
	int j;

	int wIndex = threadIdx.x + blockIdx.x * blockDim.x;
	while (wIndex < size * size) {
		i = wIndex % size;
		j = (int) (wIndex / size);
		if (i == j)
			hamiltonianMembers[wIndex] = devSpins[i + setIndex * size]
					* devMat[wIndex];
		else if (i > j)
			hamiltonianMembers[wIndex] = (double) (devSpins[i + setIndex * size]
					* devSpins[j + setIndex * size] * devMat[wIndex]);
		else
			hamiltonianMembers[wIndex] = 0;
		wIndex = wIndex + blockDim.x * gridDim.x;
	}
}

__global__ void quickSum(double* members, int size) {
	// Sum up numbers in specified range within specified pointer
	// In the end she sum will be accessible directly from pointer
	long long offset = 1;
	int wIndex;
	while (offset < size) {
		wIndex = threadIdx.x;
		while ((wIndex * 2 + 1) * offset < size) {
			members[wIndex * 2 * offset] += members[(wIndex * 2 + 1) * offset];
			wIndex = wIndex + blockDim.x;
		}
		offset *= 2;
		__syncthreads();
	}
}

double CudaAnnealing::extractHamiltonian(int index) { // Get hamiltonian from set with index
	allocateHamiltonianMembers<<<blockCount, blockSize>>>(devMat, devSpins, index, size,
			hamiltonianMembers);
	quickSum<<<1, blockSize>>>(hamiltonianMembers, size * size);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at extractEnergy");
	double out;
	checkError(
			hipMemcpy(&out, hamiltonianMembers, sizeof(double),
					hipMemcpyDeviceToHost), "memcpy energy to host");
	return out;
}

Spinset CudaAnnealing::extractSet(int index) { // Get spins from set with index
	float* hSpins = (float*) malloc(sizeof(float) * size);
	checkError(
			hipMemcpy(hSpins, &devSpins[index * size], sizeof(float) * size,
					hipMemcpyDeviceToHost), "memcpy spins to host");
	Spinset outSpins(size);
	for (int i = 0; i < size; i++)
		outSpins.SetSpin(i, hSpins[i]);
	return outSpins;
}

__device__ float meanFieldMember(const float *mat, const float *set,
		int spinIndex, int i, int size) {  // Returns /Phi_ind
	if (i != spinIndex)
		return mat[spinIndex * size + i] * set[i];
	else
		return mat[spinIndex * size + i];
}

__global__ void cudaKernelAnneal(float* mat, float* spins, int size,
		float* temp, float tempStep, float* meanFieldMembers,
		bool* proceedFlags, float proceedThreshold, int* unemptyCells,
		float linearCoef) {
	int blockId = blockIdx.x;
	int thrId = threadIdx.x;

	do {
		// Decrease temperature
		if (thrId == 0)
			temp[blockId] = temp[blockId] - tempStep;

		// Stabilize
		do {
			__syncthreads();
			// Resetting flags
			if (thrId == 0)
				proceedFlags[blockId] = false;

			for (int spinId = 0; spinId < size; ++spinId) { // Anneal every spin
				__syncthreads();

				// Mean-field member assignment
				int wIndex = thrId;

				while (wIndex < unemptyCells[spinId * (size + 1)]) {
					meanFieldMembers[wIndex + blockId * size] = meanFieldMember(
							mat, spins + blockId * size, spinId,
							unemptyCells[spinId * (size + 1) + wIndex + 1],
							size);
					wIndex = wIndex + blockDim.x;
				}
				__syncthreads();

				// Parallelized mean-field computation
				long long offset = 1;
				while (offset < unemptyCells[spinId * (size + 1)]) {
					wIndex = thrId;
					while ((wIndex * 2 + 1) * offset
							< unemptyCells[spinId * (size + 1)]) {
						meanFieldMembers[wIndex * 2 * offset + blockId * size] +=
								meanFieldMembers[(wIndex * 2 + 1) * offset
										+ blockId * size];
						wIndex = wIndex + blockDim.x;
					}
					offset *= 2;
					__syncthreads();
				}
				__syncthreads();

				// Mean-field calculation complete - write new spin and delta
				if (thrId == 0) {
					float meanField = meanFieldMembers[blockId * size];
					float old = spins[spinId + blockId * size];
					if (temp[blockId] > 0) {
						spins[spinId + blockId * size] = -1
								* tanh(meanField / temp[blockId]) * linearCoef
								+ spins[spinId + blockId * size]
										* (1 - linearCoef);
					} else if (meanField > 0)
						spins[spinId + blockId * size] = -1;
					else
						spins[spinId + blockId * size] = 1;

					if (proceedThreshold
							< fabs(old - spins[spinId + blockId * size]))
						proceedFlags[blockId] = true; // Too big delta. One more iteration needed
				}
				__syncthreads();
			}
		} while (proceedFlags[blockId]);
	} while (temp[blockId] >= 0);
}

void CudaAnnealing::anneal(float pStep, float linearCoef) {
	cudaKernelAnneal<<<blockCount, blockSize>>>(devMat, devSpins, size, devTemp,
			pStep, meanFieldMembers, continueIteration, minDiff, devUnemptyMat, linearCoef);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at cudaPull");
}
